#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>

#include <execution>
#include <iostream>
#include <iomanip>
#include <limits>
#include <optional>
#include <vector>
#include <algorithm>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include "coef.cuh"
#include "metrics.cuh"
#include "math.cuh"
#include "utils.cuh"
namespace py = pybind11;

// Debug mode macro - set to 1 to enable debug output, 0 to disable
#define DEBUG_MODE 1

template <typename T>
__global__ void findMaxAriKernel(const T *aris,
                                 // unsigned int* max_parts,
                                 T *cm_values,
                                 const int n_partitions,
                                 const int reduction_range)
{
    // Each block handles one feature comparison
    const uint64_t comp_idx = blockIdx.x;

    // Calculate start index for this feature comparison
    const uint64_t reduce_start_idx = comp_idx * reduction_range;

    // Thread-local variables for reduction
    uint64_t max_idx = UINT64_MAX;
    T max_val = -1.0f;

    // Have threads collaboratively process all partition pairs
    for (uint64_t i = threadIdx.x; i < reduction_range; i += blockDim.x)
    {
        uint64_t idx = reduce_start_idx + i;
        T val = aris[idx];

        if (val > max_val)
        {
            max_val = val;
            max_idx = i;
        }
    }

    // Shared memory for block reduction
    __shared__ typename hipcub::BlockReduce<T, 128>::TempStorage temp_storage_val;
    __shared__ typename hipcub::BlockReduce<uint64_t, 128>::TempStorage temp_storage_idx;

    // Pair-wise reduction within the block
    struct
    {
        T val;
        uint64_t idx;
    } in, out;

    in.val = max_val;
    in.idx = max_idx;

    // Find the maximum value and its index within the block
    T max_block_val = hipcub::BlockReduce<T, 128>(temp_storage_val).Reduce(in.val, hipcub::Max());

    // Only threads with the max value participate in index selection
    // (using __syncthreads to ensure all threads have computed max_block_val)
    __syncthreads();

    uint64_t selected_idx = UINT64_MAX;
    if (in.val == max_block_val)
    {
        selected_idx = in.idx;
    }

    // Get the smallest valid index of the max value
    uint64_t min_idx = hipcub::BlockReduce<uint64_t, 128>(temp_storage_idx).Reduce(selected_idx, [](uint64_t a, uint64_t b)
                                                                                { return (a == UINT64_MAX) ? b : ((b == UINT64_MAX) ? a : min(a, b)); });

    // Thread 0 writes the results
    if (threadIdx.x == 0)
    {
        cm_values[comp_idx] = max_block_val > 0.0f ? max_block_val : 0.0f;

        // Unravel the index to get partition indices
        // unsigned int m, n;
        // m = min_idx / n_partitions;
        // n = min_idx % n_partitions;

        // max_parts[comp_idx >> 1] = m;
        // max_parts[comp_idx >> 1 + 1] = n;
    }
}

// TODO: Add mode check to decide whether to do batch processing or not
template <typename T, typename R>
auto compute_coef(const py::array_t<T, py::array::c_style> &parts,
                  const size_t n_features,
                  const size_t n_partitions,
                  const size_t n_objects,
                  const bool return_parts,
                  std::optional<unsigned int> pvalue_n_perms) -> py::object
{
    // Check CUDA info
#if DEBUG_MODE
    print_cuda_device_info();
    print_cuda_memory_info();
#endif

    // Batch-computing configs, to be tuned and dynamically set based on the GPU memory size
    const uint64_t batch_n_features = 5000;
    const uint64_t batch_n_parts = n_partitions; // k from 2 to 10
    const uint64_t batch_n_feature_comp = batch_n_features * (batch_n_features - 1) / 2;
    const uint64_t batch_n_aris = batch_n_feature_comp * batch_n_parts * batch_n_parts;

    // Pre-computation
    // Check for overflow in n_feature_comp calculation
    if (n_features > 1 && n_features > UINT64_MAX / (n_features - 1))
    {
        throw std::overflow_error("Overflow in n_feature_comp calculation: n_features too large");
    }
    const uint64_t n_feature_comp = n_features * (n_features - 1) / 2;

    // Check for overflow in n_aris calculation
    if (n_feature_comp > UINT64_MAX / n_partitions)
    {
        throw std::overflow_error("Overflow in n_aris calculation: n_feature_comp * n_partitions too large");
    }
    const uint64_t temp = n_feature_comp * n_partitions;
    if (temp > UINT64_MAX / n_partitions)
    {
        throw std::overflow_error("Overflow in n_aris calculation: n_feature_comp * n_partitions * n_partitions too large");
    }
    const uint64_t n_aris = temp * n_partitions;
    const uint64_t reduction_range = n_partitions * n_partitions;

#if DEBUG_MODE
    std::cout << "\nDebug Info:" << std::endl;
    std::cout << "  n_features: " << n_features << std::endl;
    std::cout << "  n_partitions: " << n_partitions << std::endl;
    std::cout << "  n_objects: " << n_objects << std::endl;
    std::cout << "  n_feature_comp: " << n_feature_comp << std::endl;
    std::cout << "  n_aris: " << n_aris << std::endl;
    std::cout << "  batch_n_aris: " << batch_n_aris << std::endl;
#endif

    // Allocate host memory for results
#if DEBUG_MODE
    std::cout << "\nAllocating host memory..." << std::endl;
    std::cout << "  Memory before allocation: ";
    size_t before_host_mem = print_host_memory_info();
#else
    size_t before_host_mem = 0;
#endif

    std::vector<R> cm_values(n_feature_comp, -1.0f);
    std::vector<R> cm_pvalues;

    if (pvalue_n_perms.has_value())
    {
        cm_pvalues.resize(n_feature_comp, std::numeric_limits<R>::quiet_NaN());
    }

#if DEBUG_MODE
    std::cout << "  Memory after allocation: ";
    size_t after_host_mem = print_host_memory_info();
    std::cout << "  Memory used: " << (after_host_mem - before_host_mem) << " MB" << std::endl;
#endif

    // Pre-allocate device memory for the maximum batch size
    const uint64_t max_batch_feature_comp = batch_n_feature_comp;
#if DEBUG_MODE
    std::cout << "\nAllocating device memory..." << std::endl;
    std::cout << "  max_batch_feature_comp: " << max_batch_feature_comp << std::endl;
    std::cout << "  Memory before allocation: ";
    size_t before_mem = print_cuda_memory_info();
#else
    size_t before_mem = 0;
#endif

    thrust::device_vector<R> d_cm_values(max_batch_feature_comp);
    std::vector<R> batch_cm_values(max_batch_feature_comp);

#if DEBUG_MODE
    std::cout << "  Memory after allocation: ";
    size_t after_mem = print_cuda_memory_info();
    std::cout << "  Memory used: " << (before_mem - after_mem) / 1024 / 1024 << " MB" << std::endl;
#endif

    // Process ARIs in batches
    for (uint64_t batch_start = 0; batch_start < n_aris; batch_start += batch_n_aris)
    {
        // Debug - print iteration info
#if DEBUG_MODE
        std::cout << "\nProcessing batch " << (batch_start / batch_n_aris + 1) << " of "
                  << (n_aris + batch_n_aris - 1) / batch_n_aris << std::endl;
        std::cout << "  Start index: " << batch_start << std::endl;
        std::cout << "  Batch size: " << batch_n_aris << std::endl;
        std::cout << "  Memory before batch: ";
        before_mem = print_cuda_memory_info();
#endif

        // Calculate the actual batch size for this iteration
        const uint64_t current_batch_size = std::min(batch_n_aris, n_aris - batch_start);
#if DEBUG_MODE
        std::cout << "  Current batch size: " << current_batch_size << std::endl;
#endif

        try
        {
            // Compute the ARIs for this batch
            const auto d_aris = ari_core_device<T, R>(
                parts, n_features, n_partitions, n_objects, batch_start, current_batch_size);

            // Configure kernel launch parameters for this batch
            const int threadsPerBlock = 128;
            const int numBlocks = current_batch_size / (n_partitions * n_partitions);
#if DEBUG_MODE
            std::cout << "  Launching reduction kernel with " << numBlocks << " blocks, "
                      << threadsPerBlock << " threads per block" << std::endl;
#endif

            // Launch kernel to find maximum values on device for this batch
            findMaxAriKernel<R><<<numBlocks, threadsPerBlock>>>(
                thrust::raw_pointer_cast(d_aris->data()),
                thrust::raw_pointer_cast(d_cm_values.data()),
                n_partitions,
                reduction_range);

            // Check for kernel errors
            hipError_t kernelError = hipGetLastError();
            if (kernelError != hipSuccess)
            {
                throw std::runtime_error("Kernel launch failed: " + std::string(hipGetErrorString(kernelError)));
            }

            // Synchronize to ensure kernel completion
            hipError_t syncError = hipDeviceSynchronize();
            if (syncError != hipSuccess)
            {
                throw std::runtime_error("Device synchronization failed: " + std::string(hipGetErrorString(syncError)));
            }

            // Copy reduced results back to host
            thrust::copy(d_cm_values.begin(), d_cm_values.begin() + current_batch_size / (n_partitions * n_partitions),
                         batch_cm_values.begin());

            // Update the main cm_values array with the batch results
            for (uint64_t i = 0; i < current_batch_size / (n_partitions * n_partitions); ++i)
            {
                const uint64_t global_idx = batch_start / (n_partitions * n_partitions) + i;
                if (global_idx < n_feature_comp)
                {
                    cm_values[global_idx] = std::max(cm_values[global_idx], batch_cm_values[i]);
                }
            }

#if DEBUG_MODE
            std::cout << "  Memory after batch: ";
            size_t after_mem = print_cuda_memory_info();
            std::cout << "  Memory used in batch: " << (before_mem - after_mem) / 1024 / 1024 << " MB" << std::endl;
#endif
        }
        catch (const std::exception &e)
        {
            std::cerr << "\nError in batch processing:" << std::endl;
            std::cerr << "  Batch start: " << batch_start << std::endl;
            std::cerr << "  Batch size: " << current_batch_size << std::endl;
            std::cerr << "  Error: " << e.what() << std::endl;
            throw; // Re-throw to maintain error propagation
        }
    }

    // Replace -1.0f with NaN using parallel transform
    std::transform(std::execution::par,
                   cm_values.begin(), cm_values.end(), cm_values.begin(),
                   [](const R &val)
                   { return val == -1.0f ? std::numeric_limits<R>::quiet_NaN() : val; });

    // Allocate py::arrays for the results
    const auto cm_values_py = py::array_t<R>(cm_values.size(), cm_values.data());
    const auto cm_pvalues_py = pvalue_n_perms.has_value()
                                   ? py::object(py::array_t<R>(cm_pvalues.size(), cm_pvalues.data()))
                                   : py::object(py::none());

    // Return the results as a tuple
    return py::make_tuple(
        cm_values_py,
        cm_pvalues_py,
        py::object(py::none()));
}

auto example_return_optional_vectors(bool include_first,
                                     bool include_second,
                                     bool include_third) -> py::object
{
    // Example vectors
    std::optional<std::vector<float>> vec1;
    std::optional<std::vector<int>> vec2;
    std::optional<std::vector<double>> vec3;

    // Fill vectors if included
    if (include_first)
    {
        vec1 = std::vector<float>{1.0f, 2.0f, 3.0f};
    }
    if (include_second)
    {
        vec2 = std::vector<int>{4, 5, 6};
    }
    if (include_third)
    {
        vec3 = std::vector<double>{7.0, 8.0, 9.0};
    }

    // Convert to Python objects
    py::object py_vec1 = vec1.has_value() ? py::cast(vec1.value()) : py::none();
    py::object py_vec2 = vec2.has_value() ? py::cast(vec2.value()) : py::none();
    py::object py_vec3 = vec3.has_value() ? py::cast(vec3.value()) : py::none();

    // Return as tuple
    return py::make_tuple(py_vec1, py_vec2, py_vec3);
}

// Below is the explicit instantiation of the ari template function.
//
// Generally people would write the implementation of template classes and functions in the header file. However, we
// separate the implementation into a .cpp file to make things clearer. In order to make the compiler know the
// implementation of the template functions, we need to explicitly instantiate them here, so that they can be picked up
// by the linker.
template auto compute_coef<int8_t, float>(const py::array_t<int8_t, py::array::c_style> &parts,
                                          const size_t n_features,
                                          const size_t n_partitions,
                                          const size_t n_objects,
                                          const bool return_parts,
                                          std::optional<unsigned int> pvalue_n_perms) -> py::object;
