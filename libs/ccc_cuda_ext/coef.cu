#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>

#include <iostream>
#include <limits>
#include <optional>
#include <vector>
#include <algorithm>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include "coef.cuh"
#include "metrics.cuh"
#include "math.cuh"
#include "utils.cuh"
namespace py = pybind11;

template <typename T>
__global__ void findMaxAriKernel(const T* aris,
                                // unsigned int* max_parts,
                                T* cm_values,
                                const int n_partitions,
                                const int reduction_range) {
    // Each block handles one feature comparison
    const int comp_idx = blockIdx.x;

    // Calculate start index for this feature comparison
    const int reduce_start_idx = comp_idx * reduction_range;

    // Thread-local variables for reduction
    int max_idx = -1;
    T max_val = -1.0f;

    // Have threads collaboratively process all partition pairs
    for (int i = threadIdx.x; i < reduction_range; i += blockDim.x) {
        int idx = reduce_start_idx + i;
        T val = aris[idx];

        if (val > max_val) {
            max_val = val;
            max_idx = i;
        }
    }

    // Shared memory for block reduction
    __shared__ typename hipcub::BlockReduce<T, 128>::TempStorage temp_storage_val;
    __shared__ typename hipcub::BlockReduce<int, 128>::TempStorage temp_storage_idx;

    // Pair-wise reduction within the block
    struct {
        T val;
        int idx;
    } in, out;

    in.val = max_val;
    in.idx = max_idx;

    // Find the maximum value and its index within the block
    T max_block_val = hipcub::BlockReduce<T, 128>(temp_storage_val).Reduce(in.val, hipcub::Max());

    // Only threads with the max value participate in index selection
    // (using __syncthreads to ensure all threads have computed max_block_val)
    __syncthreads();

    int selected_idx = -1;
    if (in.val == max_block_val) {
        selected_idx = in.idx;
    }

    // Get the smallest valid index of the max value
    int min_idx = hipcub::BlockReduce<int, 128>(temp_storage_idx).Reduce(
        selected_idx,
        [](int a, int b) { return (a == -1) ? b : ((b == -1) ? a : min(a, b)); }
    );

    // Thread 0 writes the results
    if (threadIdx.x == 0) {
        cm_values[comp_idx] = max_block_val > 0.0f ? max_block_val : 0.0f;

        // Unravel the index to get partition indices
        // unsigned int m, n;
        // m = min_idx / n_partitions;
        // n = min_idx % n_partitions;

        // max_parts[comp_idx >> 1] = m;
        // max_parts[comp_idx >> 1 + 1] = n;
    }
}


/**
 * @brief Helper function to process and validate input numpy array
 * @param parts Input numpy array to process
 * @return Pointer to the underlying data
 */
template <typename T>
T *process_input_array(const py::array_t<T, py::array::c_style> &parts)
{
    py::buffer_info buffer = parts.request();
    if (buffer.format != py::format_descriptor<T>::format())
    {
        throw std::runtime_error("Incompatible format: expected an int array!");
    }
    if (buffer.ndim != 3)
    {
        throw std::runtime_error("Incompatible buffer dimension!");
    }
    return static_cast<T *>(buffer.ptr);
}

template <typename T>
auto compute_coef(const py::array_t<T, py::array::c_style> &parts,
                  const size_t n_features,
                  const size_t n_partitions,
                  const size_t n_objects,
                  const size_t max_k,
                  const bool return_parts,
                  std::optional<unsigned int> pvalue_n_perms) -> py::object
{
    // Pre-computation
    using parts_dtype = T;
    using out_dtype = float;
    const int n_feature_comp = n_features * (n_features - 1) / 2;
    const int n_aris = n_feature_comp * n_partitions * n_partitions;
    const auto n_elems_per_feat = n_partitions * n_objects;
    const auto reduction_range = n_partitions * n_partitions;
    // Parse the input parts
    const auto parts_ptr = process_input_array(parts);
    // Input validation
    if (!parts_ptr || n_features == 0 || n_partitions == 0 || n_objects == 0)
    {
        throw std::invalid_argument("Invalid input parameters");
    }


    // Allocate host memory for results
    std::vector<out_dtype> cm_values(n_feature_comp);
    std::vector<int32_t> max_parts(n_feature_comp * 2);
    std::vector<out_dtype> cm_pvalues;

    const size_t n_live_reductions = 0; // Number of stream groups running concurrently, we need to sync them to get partial results for cm_values

    const auto n_streams = n_partitions * n_partitions; // k * k partition aris
    // Each stream group is responsible for all ARI computations between two features
    std::vector<hipStream_t> streams(n_streams);
    for (int s = 0; s < n_streams; s++)
    {
        // Create streams, each stream is responsible for one ARI computation between two features
        CUDA_CHECK_MANDATORY(hipStreamCreate(&streams[s]));
    }

    // Compute the aris across all features and perform reduction on the go
    for (size_t range_ari_idx = 0; range_ari_idx < n_aris; range_ari_idx+=reduction_range) {
        // Allocate page-locked memory for ARI values
        const parts_dtype *h_aris;
        // Host page-locked memory
        std::vector<parts_dtype *> h_part0s(n_streams);
        std::vector<parts_dtype *> h_part1s(n_streams);
        // Device memory
        // TODO: OPTIMIZE: maybe it's better to put the whole parts arrays on the device
        std::vector<parts_dtype *> d_part0s(n_streams);
        std::vector<parts_dtype *> d_part1s(n_streams);
        // std::vector<out_dtype *> d_aris(n_streams);
        CUDA_CHECK_MANDATORY(hipHostAlloc((void **)&h_aris, reduction_range * sizeof(out_dtype), hipHostMallocDefault));
        // Copy part0 and part1 to each stream
        for (int s = 0; s < n_streams; ++s)
        {
            auto h_part0 = h_part0s[s];
            auto h_part1 = h_part1s[s];
            auto d_part0 = d_part0s[s];
            auto d_part1 = d_part1s[s];
            // auto d_ari = d_aris[s];
            // Allocate page-locked memory for part0 and part1
            CUDA_CHECK_MANDATORY(hipHostAlloc((void **)&h_part0, reduction_range * sizeof(parts_dtype), hipHostMallocDefault));
            CUDA_CHECK_MANDATORY(hipHostAlloc((void **)&h_part1, reduction_range * sizeof(parts_dtype), hipHostMallocDefault));
            CUDA_CHECK_MANDATORY(hipHostAlloc((void **)&h_aris, reduction_range * sizeof(out_dtype), hipHostMallocDefault));
            CUDA_CHECK_MANDATORY(hipMalloc((void **)&d_part0, reduction_range * sizeof(parts_dtype)));
            CUDA_CHECK_MANDATORY(hipMalloc((void **)&d_part1, reduction_range * sizeof(parts_dtype)));
            // Single ARI value per stream. We can also try one stream for all ARI values within the reduction range
            // CUDA_CHECK_MANDATORY(hipMalloc((void **)&d_aris, 1 * sizeof(out_dtype)));
            // Compute indices
            const auto feature_comp_flat_idx = range_ari_idx;
            const auto part_pair_flat_idx = s;
            uint32_t i, j;
            get_coords_from_index(n_features, feature_comp_flat_idx, &i, &j);
            uint32_t m, n;
            unravel_index(part_pair_flat_idx, n_partitions, &m, &n);
            // Copy data from parts to page-locked memory
            const auto part0_start_idx = parts_ptr + i * n_elems_per_feat + m * n_objects;
            const auto part1_start_idx = parts_ptr + j * n_elems_per_feat + n * n_objects;
            for (int k = 0; k < n_objects; ++k) {
                h_part0[k] = part0_start_idx[k];
                h_part1[k] = part1_start_idx[k];
            }
            // Copy the locked memory to the device, async
            CUDA_CHECK_MANDATORY(hipMemcpyAsync(d_part0, h_part0, reduction_range * sizeof(parts_dtype), hipMemcpyHostToDevice, streams[s]));
            CUDA_CHECK_MANDATORY(hipMemcpyAsync(d_part1, h_part1, reduction_range * sizeof(parts_dtype), hipMemcpyHostToDevice, streams[s]));
        }
        // Invoke the kernel
        for (int s = 0; s < n_streams; ++s) {
            auto d_part0 = d_part0s[s];
            auto d_part1 = d_part1s[s];
            ari_core_scalar(d_part0, d_part1, n_objects, max_k, s, streams[s], h_aris);
        }

        // Wait for all streams to finish
        for (int s = 0; s < n_streams; ++s) {
            CUDA_CHECK_MANDATORY(hipStreamSynchronize(streams[s]));
        }

        // Get the maximum ARI value and its index in array h_aris
        out_dtype max_ari = -1.0f;
        int32_t max_ari_idx = -1;
        for (int s = 0; s < n_streams; ++s) {
            if (h_aris[s] > max_ari) {
                max_ari = h_aris[s];
                max_ari_idx = s;
            }
        }
        cm_values[range_ari_idx] = max_ari;
        // Unravel the index to get partition indices
        uint32_t m, n;
        unravel_index(max_ari_idx, n_partitions, &m, &n);
        max_parts[range_ari_idx >> 1] = m;
        max_parts[range_ari_idx >> 1 + 1] = n;

        // Clean up
        for (int s = 0; s < n_streams; ++s) {
            // Destroy the stream
            CUDA_CHECK_MANDATORY(hipStreamDestroy(streams[s]));
            // Free the memory
            CUDA_CHECK_MANDATORY(hipHostFree(h_part0s[s]));
            CUDA_CHECK_MANDATORY(hipHostFree(h_part1s[s]));
            // CUDA_CHECK_MANDATORY(hipHostFree(h_aris[s]));
            CUDA_CHECK_MANDATORY(hipFree(d_part0s[s]));
            CUDA_CHECK_MANDATORY(hipFree(d_part1s[s]));
            // CUDA_CHECK_MANDATORY(hipFree(d_aris[s]));
        }
    }

    // P-valued results
    if (pvalue_n_perms.has_value()) {
        cm_pvalues.resize(n_feature_comp, std::numeric_limits<out_dtype>::quiet_NaN());
    }

    // Allocate py::arrays for the results
    // const auto max_parts_py = py::array_t<unsigned int>(max_parts.size(), max_parts.data()).reshape({n_feature_comp, 2});
    const auto cm_values_py = py::array_t<out_dtype>(cm_values.size(), cm_values.data());
    const auto cm_pvalues_py = pvalue_n_perms.has_value()
        ? py::object(py::array_t<out_dtype>(cm_pvalues.size(), cm_pvalues.data()))
        : py::object(py::none());

    // Return the results as a tuple
    return py::make_tuple(
        cm_values_py,
        cm_pvalues_py,
        py::object(py::none())
        // max_parts_py
    );
}

auto example_return_optional_vectors(bool include_first,
                                     bool include_second,
                                     bool include_third) -> py::object
{
    // Example vectors
    std::optional<std::vector<float>> vec1;
    std::optional<std::vector<int>> vec2;
    std::optional<std::vector<double>> vec3;

    // Fill vectors if included
    if (include_first)
    {
        vec1 = std::vector<float>{1.0f, 2.0f, 3.0f};
    }
    if (include_second)
    {
        vec2 = std::vector<int>{4, 5, 6};
    }
    if (include_third)
    {
        vec3 = std::vector<double>{7.0, 8.0, 9.0};
    }

    // Convert to Python objects
    py::object py_vec1 = vec1.has_value() ? py::cast(vec1.value()) : py::none();
    py::object py_vec2 = vec2.has_value() ? py::cast(vec2.value()) : py::none();
    py::object py_vec3 = vec3.has_value() ? py::cast(vec3.value()) : py::none();

    // Return as tuple
    return py::make_tuple(py_vec1, py_vec2, py_vec3);
}

// Below is the explicit instantiation of the ari template function.
//
// Generally people would write the implementation of template classes and functions in the header file. However, we
// separate the implementation into a .cpp file to make things clearer. In order to make the compiler know the
// implementation of the template functions, we need to explicitly instantiate them here, so that they can be picked up
// by the linker.
template auto compute_coef<int8_t>(const py::array_t<int8_t, py::array::c_style> &parts,
                                const size_t n_features,
                                const size_t n_partitions,
                                const size_t n_objects,
                                const size_t max_k,
                                const bool return_parts,
                                std::optional<unsigned int> pvalue_n_perms) -> py::object;
