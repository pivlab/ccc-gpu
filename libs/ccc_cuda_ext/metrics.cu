#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cub/block/block_load.cuh>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/extrema.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <iostream>
#include <cmath>
#include <assert.h>
#include "metrics.cuh"
#include "utils.cuh"

namespace py = pybind11;

/**
 * Future optimizations
 * 1. use narrower data types
 * 2. optimized on locality
 * 3. use warp-level reduction
 */

/**
 * Future optimizations
 * 1. GPU memory is not enough to store the partitions -> split the partitions into smaller chunks
 *    and do stream processing
 * 2.
 */

/**
 * @brief Unravel a flat index to the corresponding 2D indicis
 * @param[in] flat_idx The flat index to unravel
 * @param[in] num_cols Number of columns in the 2D array
 * @param[out] row Pointer to the row index
 * @param[out] col Pointer to the column index
 */
__device__ __host__ inline void unravel_index(int flat_idx, int num_cols, int *row, int *col)
{
    // change int to uint32_t
    *row = flat_idx / num_cols; // Compute row index
    *col = flat_idx % num_cols; // Compute column index
}

/**
 * @brief Given the number of objects and an index, this function calculates
 *        the coordinates in a symmetric matrix from a flat index.
 *        For example, if there are n_obj objects (such as genes), a condensed
 *        1D array can be created with pairwise comparisons between these
 *        objects, which corresponds to a symmetric 2D matrix. This function
 *        calculates the 2D coordinates (x, y) in the symmetric matrix that
 *        corresponds to the given flat index.
 *
 * @param[in] n_obj The total number of objects (i.e., the size of one dimension
 *                  of the square symmetric matrix).
 * @param[in] idx The flat index from the condensed pairwise array.
 * @param[out] x Pointer to the calculated row coordinate in the symmetric matrix.
 * @param[out] y Pointer to the calculated column coordinate in the symmetric matrix.
 */
__device__ __host__ inline void get_coords_from_index(int n_obj, int idx, uint32_t *x, uint32_t *y)
{
    // Use int64_t to prevent overflow in intermediate calculations
    int64_t n_obj_64 = static_cast<int64_t>(n_obj);
    int64_t idx_64 = static_cast<int64_t>(idx);

    // Calculate 'b' using 64-bit arithmetic
    int64_t b = 1 - 2 * n_obj_64;

    // Calculate discriminant using 64-bit arithmetic
    // Use double for floating point to maintain precision
    double b_squared = static_cast<double>(b) * b;
    double idx_term = 8.0 * static_cast<double>(idx_64);
    double discriminant = b_squared - idx_term;

    // Calculate x using double precision
    double x_float = (-b - sqrt(discriminant)) / 2.0;

    // Floor and convert to uint32_t, with bounds checking
    int64_t x_64 = static_cast<int64_t>(floor(x_float));
    if (x_64 < 0 || x_64 > UINT32_MAX)
    {
        // Handle error condition - could throw error or set to max/min value
        *x = 0;
        *y = 0;
        return;
    }
    *x = static_cast<uint32_t>(x_64);

    // Calculate y using 64-bit arithmetic to prevent overflow
    int64_t y_term1 = idx_64;
    int64_t y_term2 = x_64 * (b + x_64 + 2) / 2;
    int64_t y_64 = y_term1 + y_term2 + 1;

    // Bounds checking for y
    if (y_64 < 0 || y_64 > UINT32_MAX)
    {
        // Handle error condition
        *x = 0;
        *y = 0;
        return;
    }
    *y = static_cast<uint32_t>(y_64);
}

/**
 * @brief Compute the contingency matrix for two partitions using shared memory
 * @param[in] part0 Pointer to the first partition array, global memory
 * @param[in] part1 Pointer to the second partition array, global memory
 * @param[in] n_objs Number of elements in each partition array
 * @param[out] shared_cont_mat Pointer to shared memory for storing the contingency matrix
 * @param[in] k Maximum number of clusters (size of contingency matrix is k x k)
 */
__device__ void get_contingency_matrix(int *part0, int *part1, int n_objs, int *shared_cont_mat, int k)
{
    const int tid = threadIdx.x;
    const int n_block_threads = blockDim.x;
    const int cont_mat_size = k * k;

    // Initialize shared memory
    if (tid < cont_mat_size)
    {
        shared_cont_mat[tid] = 0;
    }
    __syncthreads();

#pragma unroll
    for (int i = tid; i < n_objs; i += n_block_threads)
    {
        // Directly load row/col info from global memory into registers, no need to load into shared memory
        const int row = part0[i];
        const int col = part1[i];

        // Add bounds checking
        // assert(row >= 0 && row < k && col >= 0 && col < k);
        // OPT: can we use shared memory to avoid atomicAdd?
        atomicAdd(&shared_cont_mat[row * k + col], 1);
    }
    // __syncthreads();
}

/**
 * @brief CUDA device function to compute the pair confusion matrix
 * @param[in] contingency Pointer to the contingency matrix
 * @param[in] sum_rows Pointer to the sum of rows in the contingency matrix
 * @param[in] sum_cols Pointer to the sum of columns in the contingency matrix
 * @param[in] n_objs Number of objects in each partition
 * @param[in] k Number of clusters (assuming k is the max of clusters in part0 and part1)
 * @param[out] C Pointer to the output pair confusion matrix (2x2)
 */
__device__ void get_pair_confusion_matrix(
    const int *__restrict__ contingency,
    int *sum_rows,
    int *sum_cols,
    const int n_objs,
    const int k,
    int *C)
{
    // TODO: use block-level reduction

    const int tid = threadIdx.x;
    const int n_block_threads = blockDim.x;

    // Initialize sum_rows and sum_cols
    for (int i = tid; i < k; i += n_block_threads)
    {
        sum_rows[i] = 0;
        sum_cols[i] = 0;
    }
    __syncthreads();

    // Compute sum_rows and sum_cols
    for (int i = tid; i < k * k; i += n_block_threads)
    {
        int row = i / k;
        int col = i % k;
        int val = contingency[i];
        atomicAdd(&sum_cols[col], val);
        atomicAdd(&sum_rows[row], val);
    }
    __syncthreads();

    // Compute sum_squares
    int sum_squares;
    if (tid == 0)
    {
        sum_squares = 0;
        for (int i = 0; i < k * k; ++i)
        {
            sum_squares += (contingency[i] * contingency[i]);
        }
    }
    __syncthreads();

    // Use different warps to compute C[1,1], C[0,1], C[1,0], and C[0,0]
    if (tid == 0)
    {
        C[3] = sum_squares - n_objs; // C[1,1]

        int temp = 0;
        for (int i = 0; i < k; ++i)
        {
            for (int j = 0; j < k; ++j)
            {
                temp += (contingency[i * k + j]) * sum_cols[j];
            }
        }
        C[1] = temp - sum_squares; // C[0,1]

        temp = 0;
        for (int i = 0; i < k; ++i)
        {
            for (int j = 0; j < k; ++j)
            {
                temp += (contingency[j * k + i]) * sum_rows[j];
            }
        }
        C[2] = temp - sum_squares; // C[1,0]

        C[0] = n_objs * n_objs - C[1] - C[2] - sum_squares; // C[0,0]
    }
}

/**
 * @brief Main ARI kernel. Now only compare a pair of ARIs
 * @param parts Device pointer to the 3D Array of partitions with shape of (n_features, n_parts, n_objs)
 * @param n_aris Number of ARIs to compute
 * @param n_features Number of features
 * @param n_parts Number of partitions of each feature
 * @param n_objs Number of objects in each partitions
 * @param n_elems_per_feat Number of elements for each feature, i.e., part[i].x * part[i].y
 * @param n_part_mat_elems Number of elements in the square partition matrix
 * @param k The max value of cluster number + 1
 * @param out Output array of ARIs
 */
// TODO: Parameterize the int type to allow using narrower int types for memory efficiency
extern "C" __global__ void ari_kernel(int *parts,
                                      const int n_aris,
                                      const int n_features,
                                      const int n_parts,
                                      const int n_objs,
                                      const int n_elems_per_feat,
                                      const int n_part_mat_elems,
                                      const int k,
                                      float *out)
{
    /*
     * Step 0: Compute shared memory addresses
     */
    extern __shared__ int shared_mem[];
    int *s_contingency = shared_mem;               // k * k elements
    int *s_sum_rows = s_contingency + (k * k);     // k elements
    int *s_sum_cols = s_sum_rows + k;              // k elements
    int *s_pair_confusion_matrix = s_sum_cols + k; // 4 elements

    /*
     * Step 1: Each thead, unravel flat indices and load the corresponding data into shared memory
     */
    // each block is responsible for one ARI computation
    int ari_block_idx = blockIdx.x;
    // obtain the corresponding parts and unique counts
    int feature_comp_flat_idx = ari_block_idx / n_part_mat_elems; // flat comparison pair index for two features
    int part_pair_flat_idx = ari_block_idx % n_part_mat_elems;    // flat comparison pair index for two partitions of one feature pair
    uint32_t i, j;

    // Unravel the feature indices
    // For example, if n_features = 3, n_feature_comp = n_features * (n_features - 1) / 2 = 3
    // The feature indices of the pair being compared are (0, 1), (0, 2), (1, 2)
    // i.e., the pairs being compared are feature0-feature1, feature0-feature2, feature1-feature2
    // The range of the flattened index is [0, n_feature_comp - 1] = [0, 2]
    // Given the flat index, we compute the corresponding feature indices
    get_coords_from_index(n_features, feature_comp_flat_idx, &i, &j);
    // assert(i < n_features && j < n_features);
    // assert(i >= 0 && j >= 0);

    // Unravel the partition indices within the feature pair
    // For example, if n_parts = 3, n_part_mat_elems = n_parts * n_parts = 9
    // The partition indices of the pair being compared are (0, 1), (0, 2), (1, 0), (1, 1), (1, 2), (2, 0), (2, 1), (2, 2)
    // i.e., the pairs being compared are part0-part1, part0-part2, part1-part0, part1-part1, part1-part2, part2-part0, part2-part1, part2-part2
    // The range of the flattened index is [0, n_part_mat_elems - 1] = [0, 8]
    // Given the flat index, we compute the corresponding partition indices
    int m, n;
    unravel_index(part_pair_flat_idx, n_parts, &m, &n);
    // Make pointers to select the partitions from `parts` and unique counts for the feature pair
    // Todo: Use int4*?
    // Prefix `t_` for data hold by a thread
    int *t_data_part0 = parts + i * n_elems_per_feat + m * n_objs;
    int *t_data_part1 = parts + j * n_elems_per_feat + n * n_objs;

    // Check on categorical partition marker, if the first object of either partition is -1 (actually all the objects are -1),
    // then skip the computation for this feature pair. The final coef output will still have a slot for this pair, with a default value of -1.
    if (t_data_part0[0] == -1 || t_data_part1[0] == -1)
    {
        return;
    }

    /*
     * Step 2: Compute contingency matrix within the block
     */
    // shared mem address for the contingency matrix
    // int *s_contingency = shared_mem + 2 * n_objs;
    get_contingency_matrix(t_data_part0, t_data_part1, n_objs, s_contingency, k);

    /*
     * Step 3: Construct pair confusion matrix
     */
    get_pair_confusion_matrix(s_contingency, s_sum_rows, s_sum_cols, n_objs, k, s_pair_confusion_matrix);

    /*
     * Step 4: Compute ARI and write to global memory
     */
    if (threadIdx.x == 0)
    {
        float tn = s_pair_confusion_matrix[0];
        float fp = s_pair_confusion_matrix[1];
        float fn = s_pair_confusion_matrix[2];
        float tp = s_pair_confusion_matrix[3];
        float ari = 0.0f;
        if (fn == 0 && fp == 0)
        {
            ari = 1.0f;
        }
        else
        {
            ari = 2.0f * (tp * tn - fn * fp) / ((tp + fn) * (fn + tn) + (tp + fp) * (fp + tn));
        }
        out[ari_block_idx] = ari;
    }
    __syncthreads();
}

/**
 * @brief Helper function to process and validate input numpy array
 * @param parts Input numpy array to process
 * @return Pointer to the underlying data
 */
template <typename T>
T *process_input_array(const py::array_t<T, py::array::c_style> &parts)
{
    py::buffer_info buffer = parts.request();
    if (buffer.format != py::format_descriptor<T>::format())
    {
        throw std::runtime_error("Incompatible format: expected an int array!");
    }
    if (buffer.ndim != 3)
    {
        throw std::runtime_error("Incompatible buffer dimension!");
    }
    return static_cast<T *>(buffer.ptr);
}

/**
 * @brief Internal lower-level ARI computation, returns a pointer to the ARI values on the device
 * @param parts pointer to the 3D Array of partitions with shape of (n_features, n_parts, n_objs)
 * @throws std::invalid_argument if "parts" is invalid
 * @return std::unique_ptr to thrust device vector containing ARI values with type R
 */
template <typename T, typename R>
auto ari_core_device(const T *parts,
                     const size_t n_features,
                     const size_t n_parts,
                     const size_t n_objs) -> std::unique_ptr<thrust::device_vector<R>>
{
    /*
     * Show debugging and device information
     */
    // printf("Max shared memory per block: %zu bytes\n", get_max_shared_memory_per_block());

    // Input validation
    if (!parts || n_features == 0 || n_parts == 0 || n_objs == 0)
    {
        throw std::invalid_argument("Invalid input parameters");
    }

    /*
     * Pre-computation
     */
    using parts_dtype = T;
    using out_dtype = R;
    const auto n_feature_comp = n_features * (n_features - 1) / 2;
    const auto n_aris = n_feature_comp * n_parts * n_parts;

    /*
     * Memory Allocation
     */
    // Create device vectors using unique_ptr
    const auto n_elems = n_features * n_parts * n_objs;
    auto d_parts = std::make_unique<thrust::device_vector<parts_dtype>>(parts, parts + n_elems);
    auto d_out = std::make_unique<thrust::device_vector<out_dtype>>(n_aris, 0.0f);

    // Define shared memory size for each block
    // Pre-compute the max value of the partitions
    // TODO: Each block could compute the max value of the partition pairs to eliminate this global reduction
    // Also, potentially we can spare some smem for better occupancy. But the issue is we need to dynamically allocate smem for each block
    const auto k = thrust::reduce(d_parts->begin(), d_parts->end(), -1, thrust::maximum<parts_dtype>()) + 1;
    const auto sz_parts_dtype = sizeof(parts_dtype);
    // Compute shared memory size
    // FIXME: Partition pair size should be fixed. Stream processing should be used for large input
    // NOTE: Use global memory to fix the issue for now and then optimize with shared memory
    // auto s_mem_size = 2 * n_objs * sz_parts_dtype;  // For the partition pair to be compared
    auto s_mem_size = 0;
    s_mem_size += k * k * sz_parts_dtype; // For contingency matrix
    s_mem_size += 2 * k * sz_parts_dtype; // For the internal sum arrays, FIXME: should be fixed?
    s_mem_size += 4 * sz_parts_dtype;     // For the pair confusion matrix

    // Check if shared memory size exceeds device limits
    auto [is_valid, message] = check_shared_memory_size(s_mem_size);
    if (!is_valid)
    {
        throw std::runtime_error(message);
    }

    /*
     * Launch the kernel
     */
    // Each logical block is responsible for one ARI computation
    const auto grid_size = n_aris;
    // Todo: change block_size to template parameter for performance tuning
    // For now, with 128 threads per block, we have 4 warps to compute the 4 elements of the confusion matrix.
    // Future optimizations should consider how to reduce the number of idle warps.
    const auto block_size = 128;
    // Launch the kernel
    ari_kernel<<<grid_size, block_size, s_mem_size>>>(
        thrust::raw_pointer_cast(d_parts->data()),
        n_aris,
        n_features,
        n_parts,
        n_objs,
        n_parts * n_objs,
        n_parts * n_parts,
        k,
        thrust::raw_pointer_cast(d_out->data()));
    // Return the device vector
    return d_out;
}

/**
 * @brief Overloaded ari_core_device function. Takes a numpy.ndarray as input
 * @param parts 3D Numpy.NDArray of partitions with shape of (n_features, n_parts, n_objs)
 * @throws std::invalid_argument if "parts" is invalid
 * @return std::unique_ptr to thrust device vector containing ARI values
 */
template <typename T, typename R>
auto ari_core_device(const py::array_t<int, py::array::c_style> &parts,
                     const size_t n_features,
                     const size_t n_parts,
                     const size_t n_objs) -> std::unique_ptr<thrust::device_vector<R>>
{
    const auto parts_ptr = process_input_array(parts);
    return ari_core_device<T, R>(parts_ptr, n_features, n_parts, n_objs);
}

/**
 * @brief Internal lower-level ARI computation
 * @param parts pointer to the 3D Array of partitions with shape of (n_features, n_parts, n_objs)
 * @throws std::invalid_argument if "parts" is invalid
 * @return std::vector<float> ARI values for each pair of partitions stored in host memory
 */
template <typename T>
auto ari_core_host(const T *parts,
                   const size_t n_features,
                   const size_t n_parts,
                   const size_t n_objs) -> std::vector<float>
{
    /*
     * Pre-computation
     */
    using parts_dtype = T;
    using out_dtype = float;
    const auto n_feature_comp = n_features * (n_features - 1) / 2;
    const auto n_aris = n_feature_comp * n_parts * n_parts;

    /*
     * Memory Allocation
     */
    // Allocate host memory
    thrust::host_vector<out_dtype> h_out(n_aris);
    // thrust::host_vector<parts_dtype> h_parts_pairs(n_aris * 2 * n_objs);

    // Call the device function ari_core_device
    auto d_out = ari_core_device<parts_dtype, out_dtype>(parts, n_features, n_parts, n_objs);

    // Copy data back to host using -> operator since d_out is a unique_ptr
    thrust::copy(d_out->begin(), d_out->end(), h_out.begin());

    // Copy data to std::vector
    std::vector<out_dtype> res(n_aris);
    thrust::copy(h_out.begin(), h_out.end(), res.begin());

    // Return the ARI values
    return res;
}

/**********************
  API Implementations
 **********************/

/**
 * @brief API exposed to Python for computing ARI using CUDA upon a 3D Numpy NDArray of partitions
 * @param parts 3D Numpy.NDArray of partitions with shape of (n_features, n_parts, n_objs)
 * @throws std::invalid_argument if "parts" is invalid
 * @return std::vector<float> All ARI values for each pair of partitions
 */
template <typename T>
auto ari(const py::array_t<T, py::array::c_style> &parts,
         const size_t n_features,
         const size_t n_parts,
         const size_t n_objs) -> std::vector<float>
{
    const auto parts_ptr = process_input_array(parts);
    return ari_core_host(parts_ptr, n_features, n_parts, n_objs);
}

/**
 * @brief API exposed to Python for computing ARI using CUDA upon a 3D Numpy NDArray of partitions
 * @param parts 3D Numpy.NDArray of partitions with shape of (n_features, n_parts, n_objs)
 * @throws std::invalid_argument if "parts" is invalid
 * @return std::vector<float> Reduced(max) ARI value for each pair of partitions
 */
template <typename T>
auto ari_reduced(const py::array_t<T, py::array::c_style> &parts,
                 const size_t n_features,
                 const size_t n_parts,
                 const size_t n_objs) -> std::vector<float>
{
    const auto parts_ptr = process_input_array(parts);
    throw std::logic_error("Function not yet implemented");
}

// Below is the explicit instantiation of the ari template function.
//
// Generally people would write the implementation of template classes and functions in the header file. However, we
// separate the implementation into a .cpp file to make things clearer. In order to make the compiler know the
// implementation of the template functions, we need to explicitly instantiate them here, so that they can be picked up
// by the linker.

// Used for external python testing
template auto ari<int>(
    const py::array_t<int, py::array::c_style> &parts,
    const size_t n_features,
    const size_t n_parts,
    const size_t n_objs) -> std::vector<float>;

// Used for internal c++ testing
template auto ari_core_host<int>(
    const int *parts,
    const size_t n_features,
    const size_t n_parts,
    const size_t n_objs) -> std::vector<float>;

// Used in the coef API
template auto ari_core_device<int, float>(
    const py::array_t<int, py::array::c_style> &parts,
    const size_t n_features,
    const size_t n_parts,
    const size_t n_objs) -> std::unique_ptr<thrust::device_vector<float>>;
